
#include <hip/hip_runtime.h>
#include<stdio.h>

// __global__: 告诉编译器这个是可以在设备上执行的核函数
__global__ void hello_world(void) {
  printf("GPU: Hello world!\n");
}

int main(int argc, char **argv) {
  printf("CPU: Hello world!\n");
  hello_world<<<1,10>>>();  // <<<...>>>: 对设备进行配置的参数
  hipDeviceReset();  // if no this line ,it can not output hello world from gpu (隐式同步)
  return 0;
}

/*
 * 知识点:
 * 1. CPU和GPU执行程序是异步的, 核函数调用后会立刻到主机线程继续执行
 *    而不管GPU端核函数是否执行完毕
 * 2. 一般CUDA程序分为如下步骤: 
 *    2.1 分配GPU内存
 *    2.2 拷贝内存到设备
 *    2.3 调用CUDA内核函数来执行计算
 *    2.4 把计算完成的数据拷贝回主机端
 *    2.5 内存销毁
 */